#include "hip/hip_runtime.h"
#include <string>
#include <iostream>
#include <stdio.h>
#include <fstream>
#include <math.h>
#include <algorithm>
#include <stdlib.h>

#define BLOCK_X 8
#define BLOCK_Y 8
#define BLOCK_Z 4

using namespace std;

string output_direc = "./out";
string filename = "bif";
string logname;

const int Q = 19, NX = 64, NY = 83, NZ = 32,REPEAT=4400,time_save=4400;//D3Q19
const float CH = 0.000248925f, C_rho = 998.2f, C_U = 0.24159041f;
float *h_ux, *h_uy, *h_uz, *h_rho,*h_inletx,*h_inlety,*h_outletx,*h_outlety,*h_meax,*h_meay, *d_dst, *d_scr,*h_dst, *h_scr, *d_tmp,*d_ux,*d_uy,*d_uz,*d_rho,*d_meax1,*d_meay1,*d_inletx1,*d_inlety1,*d_outletx1,*d_outlety1;
int *h_geo,*d_geo1,*h_index,*d_indexdev;
float C_pre = C_rho*C_U*C_U;
int bx=1+(NX-1)/BLOCK_X,by=1+(NY-1)/BLOCK_Y,bz=1+(NZ-1)/BLOCK_Z;
int NLATTICE=0;
float wi[19] = { 1.0f / 3.0f, 1.0f / 18.0f,1.0f / 18.0f,1.0f / 18.0f,1.0f / 18.0f,1.0f / 18.0f,1.0f / 18.0f,1.0f / 36.0f,1.0f / 36.0f,1.0f / 36.0f,1.0f / 36.0f,1.0f / 36.0f,1.0f / 36.0f,1.0f / 36.0f,1.0f / 36.0f,1.0f / 36.0f,1.0f / 36.0f,1.0f / 36.0f,1.0f / 36.0f };
texture<float,hipTextureType1D,hipReadModeElementType> d_meax;
texture<float,hipTextureType1D,hipReadModeElementType> d_meay;
texture<int,hipTextureType1D,hipReadModeElementType> d_index;
texture<float,hipTextureType1D,hipReadModeElementType> d_inletx;
texture<float,hipTextureType1D,hipReadModeElementType> d_inlety;
texture<float,hipTextureType1D,hipReadModeElementType> d_outletx;
texture<float,hipTextureType1D,hipReadModeElementType> d_outlety;
texture<int,hipTextureType1D,hipReadModeElementType> d_geo;

void geo_pre() {
	//not useful=0,ghost cells=-1, wall=1,left end=2,right end=3,fluid=4,augmentation=5
	int minn4, minx, miny, minz, minn8, ind, ind2, i, j, k, tmp;
	int ***flag;
	FILE *file;
	flag = (int ***)malloc(NX * sizeof(*flag));

	for (int x = 0; x < NX; x++) {
		flag[x] = (int **)malloc(NY * sizeof(**flag));
		for (int y = 0; y < NY; y++) {
			flag[x][y] = (int *)malloc(NZ * sizeof(***flag));
		}
	}

	file=fopen("./geo.txt", "r");
	for (int z = 0; z < NZ; z++) {
		for (int y = 0; y < NY; y++) {
			for (int x = 0; x < NX; x++) {
				ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				fscanf(file, "%d ", &tmp);
				h_geo[ind] = tmp;
				flag[x][y][z] = tmp;
			}
		}
	}
	fclose(file);

	int y;
	for (int x = 1; x < NX - 1; x++) {
		for (int z = 1; z < NZ - 1; z++) {
			y = 0;
			ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			h_geo[ind] = 0;
			y = NY - 1;
			ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			h_geo[ind] = 0;
		}
	}
	

	//distance transform, fluid=4
	for (int t = 0; t < 3; t++) {
		for (int x = 1; x < NX - 1; x++) {
			for (int y = 2; y < NY - 2; y++) {
				for (int z = 1; z < NZ - 1; z++) {
					ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
					minx = min(flag[x + 1][y][z], flag[x - 1][y][z]);
					miny = min(flag[x][y - 1][z], flag[x][y + 1][z]);
					minz = min(flag[x][y][z - 1], flag[x][y][z + 1]);
					minn4 = min(minx, miny);
					minn8 = min(minn4, minz);
					h_geo[ind] = h_geo[ind] + minn8;
				}
			}
		}
	}

	//distance transform, left end=2
	for (int x = 1; x < NX - 1; x++) {
		for (int z = 1; z < NZ - 1; z++) {
			y = 1;
			ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			y=2;
			ind2= (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			h_geo[ind]=0;
			if(h_geo[ind2]==1)h_geo[ind]=1;
			if(h_geo[ind2]==4)h_geo[ind]=2;
		}
	}
	
	//distance transform, right end=3
	for (int t = 0; t < 2; t++) {
		for (int x = 1; x < NX - 1; x++) {
			for (int z = 1; z < NZ - 1; z++) {
				y = NY - 2;
				ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				y=NY-3;
				ind2= (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				h_geo[ind]=0;
				if(h_geo[ind2]==1)h_geo[ind]=1;
				if(h_geo[ind2]==4)h_geo[ind]=3;
			}
		}
	}

	free(flag);
	
	for (int z = 1; z < NZ - 1; z++) {
		for (int y = 1; y < NY - 1; y++) {
			for (int x = 1; x < NX - 1; x++) {
				ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				if (h_geo[ind] == 1 ) {
					//1
					i = x + 1;
					j = y;
					k = z;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//2
					i = x - 1;
					j = y;
					k = z;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//3
					i = x;
					j = y + 1;
					k = z;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//4
					i = x;
					j = y - 1;
					k = z;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//5
					i = x;
					j = y;
					k = z + 1;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//6
					i = x;
					j = y;
					k = z - 1;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//7
					i = x + 1;
					j = y + 1;
					k = z;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//8
					i = x + 1;
					j = y - 1;
					k = z;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//9
					i = x - 1;
					j = y + 1;
					k = z;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//10
					i = x - 1;
					j = y - 1;
					k = z;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//11
					i = x + 1;
					j = y;
					k = z + 1;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//12
					i = x + 1;
					j = y;
					k = z - 1;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//13
					i = x - 1;
					j = y;
					k = z + 1;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//14
					i = x - 1;
					j = y;
					k = z - 1;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//15
					i = x;
					j = y + 1;
					k = z + 1;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//16
					i = x;
					j = y - 1;
					k = z + 1;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//17
					i = x;
					j = y + 1;
					k = z - 1;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
					//18
					i = x;
					j = y - 1;
					k = z - 1;
					ind2 = (i / BLOCK_X + j / BLOCK_Y*bx + k / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + i%BLOCK_X + j%BLOCK_Y*BLOCK_X + k%BLOCK_Z*BLOCK_X*BLOCK_Y;
					if (h_geo[ind2] == 0)h_geo[ind2] = -1;
				}
			}
		}
	}

	for (int z = 0; z < NZ; z++) {
		for (int y = 0; y < NY; y++) {
			for (int x = 0; x < NX; x++) {
				ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				if (h_geo[ind] != 0) {
					h_index[ind] = NLATTICE;
					NLATTICE++;
				}
				else h_index[ind] = -1;
			}
		}
	}
}

void read_vel() {
	int ind, ind1, z = NZ/2;
	FILE *file, *file1;
	float tmp;
	
	/**
	file=fopen("./measurements.txt", "r");	
	for (int y = 0; y < NY; y++) {
		for (int x = 0; x < NX; x++) {
			ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			ind1 = x + y*NX;
			fscanf(file, "%f ", &tmp);
			if (h_geo[ind] == 4) {
				h_meay[ind1] = tmp;
			}
			else {
				h_meay[ind1] = 0;
			}
		}
	}
	
	for (int y = 0; y < NY; y++) {
		for (int x = 0; x < NX; x++) {
			ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			ind1 = x + y*NX;
			fscanf(file, "%f ", &tmp);

			if (h_geo[ind] == 4) {
				h_meax[ind1] = tmp;
			}
			else {
				h_meax[ind1] = 0;
			}

		}
	}
	fclose(file);
	**/
	
	int y = 1;
	file1=fopen("./bc.txt", "r");
	for (int z = 0; z < NZ; z++) {
		for (int x = 0; x < NX; x++) {
			ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			ind1 = x + z*NX;
			fscanf(file1, "%f ", &tmp);
			if (h_geo[ind] == 2) {
				h_inlety[ind1] = tmp;
			}
			else {
				h_inlety[ind1] = 0;
			}
			h_inletx[ind1]=0;
		}
	}
	
	y = NY - 2;
	for (int z = 0; z < NZ; z++) {
		for (int x = 0; x < NX; x++) {
			ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			ind1 = x + z*NX;
			fscanf(file1, "%f ", &tmp);
			if (h_geo[ind] == 3) {
				h_outlety[ind1] = tmp;
			}
			else {
				h_outlety[ind1] = 0;
			}
			h_outletx[ind1]=0;
		}
	}
	fclose(file1);
}

void initialize() {
	int ind,idx,ind1;
	float feq[19];
	float  tmp_rho, tmp_ux, tmp_uy, tmp_uz;
	
	for (int z = 0; z < NZ; z++) {
		for (int y = 0; y < NY; y++) {
			for (int x = 0; x < NX; x++) {
				ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				idx=h_index[ind];
				if(idx>=0){
					h_ux[idx] = 0.0f;
					h_uy[idx] = 0.0f;
					h_uz[idx] = 0.0f;
					h_rho[idx] = 1.0f;
				}
			}
		}
	}
	
	int y=1;
	for (int x = 0; x<NX; x++) {
		for (int z = 0; z<NZ; z++) {
			ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=h_index[ind];				
			ind1 = x + z*NX;
			if(idx>=0){
				h_ux[idx] =0.0f;
				h_uy[idx] =h_inlety[ind1];
			}
		}
	}
	
	y=NY-2;
	for (int x = 0; x<NX; x++) {
		for (int z = 0; z<NZ; z++) {
			ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=h_index[ind];				
			ind1 = x + z*NX;
			if(idx>=0){
				h_ux[idx] =0.0f;
				h_uy[idx] =h_outlety[ind1];
			}
		}
	}
	/**
	int z=NZ/2;
	for (int x = 0; x<NX; x++) {
		for (int y = 0; y<NY; y++) {
			ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=h_index[ind];
			ind1 = x + y*NX;			
			if(h_geo[ind]==4){
				h_ux[idx] = h_meax[ind1];
				h_uy[idx] = h_meay[ind1];
			}
		}
	}
	**/
	for (int x=0; x < NX; x++) {
		for (int y=0; y < NY; y++) {
			for (int z=0; z < NZ ; z++) {
				ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				idx=h_index[ind];				
				if(idx>=0){
					tmp_rho = h_rho[idx];
					tmp_ux = h_ux[idx];
					tmp_uy = h_uy[idx];
					tmp_uz = h_uz[idx];

					feq[0] = tmp_rho/3.0f * (1.0f - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uy*tmp_uy -1.5f* tmp_uz*tmp_uz);
					feq[1] = tmp_rho /18.0f * (1.0f + 3.0f* tmp_ux + 3.0f*tmp_ux*tmp_ux -1.5f* tmp_uy*tmp_uy -1.5f* tmp_uz*tmp_uz);
					feq[2] = tmp_rho /18.0f * (1.0f - 3.0f* tmp_ux + 3.0f*tmp_ux*tmp_ux -1.5f* tmp_uy*tmp_uy -1.5f* tmp_uz*tmp_uz);
					feq[3] = tmp_rho /18.0f * (1.0f + 3.0f* tmp_uy + 3.0f*tmp_uy*tmp_uy - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uz*tmp_uz);
					feq[4] = tmp_rho /18.0f * (1.0f - 3.0f* tmp_uy + 3.0f*tmp_uy*tmp_uy - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uz*tmp_uz);
					feq[5] = tmp_rho /18.0f * (1.0f + 3.0f* tmp_uz + 3.0f*tmp_uz*tmp_uz - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uy*tmp_uy);
					feq[6] = tmp_rho /18.0f* (1.0f - 3.0f* tmp_uz + 3.0f*tmp_uz*tmp_uz - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uy*tmp_uy);
					feq[7] = tmp_rho /36.0f* (1.0f + 3.0f* (tmp_ux + tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy + 9.0f*tmp_ux*tmp_uy -1.5f* tmp_uz*tmp_uz);
					feq[8] = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_ux - tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy - 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
					feq[9] = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uy - tmp_ux) + 3.0f* tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy - 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
					feq[10] = tmp_rho /36.0f * (1.0f - 3.0f* (tmp_ux + tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy + 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
					feq[11] = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_ux + tmp_uz) + 3.0f* tmp_ux*tmp_ux + 3.0f*tmp_uz*tmp_uz + 9.0f*tmp_ux*tmp_uz-1.5f* tmp_uy*tmp_uy);
					feq[12] = tmp_rho /36.0f* (1.0f + 3.0f* (tmp_ux - tmp_uz) + 3.0f* tmp_ux*tmp_ux + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_ux*tmp_uz-1.5f* tmp_uy*tmp_uy);
					feq[13] = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uz - tmp_ux) + 3.0f* tmp_ux*tmp_ux + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_ux*tmp_uz-1.5f* tmp_uy*tmp_uy);
					feq[14] = tmp_rho /36.0f * (1.0f - 3.0f* (tmp_ux + tmp_uz) + 3.0f* tmp_ux*tmp_ux + 3.0*tmp_uz*tmp_uz + 9.0f*tmp_ux*tmp_uz -1.5f* tmp_uy*tmp_uy);
					feq[15] = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uy + tmp_uz) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz + 9.0f*tmp_uy*tmp_uz- 1.5f*tmp_ux*tmp_ux);
					feq[16] = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uz - tmp_uy) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
					feq[17] = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uy - tmp_uz) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
					feq[18] = tmp_rho /36.0f * (1.0f - 3.0f* (tmp_uy + tmp_uz) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz + 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
			
					for (int k = 0; k<Q; k++) {
						h_dst[NLATTICE*k+idx] = feq[k];
						h_scr[NLATTICE*k+idx] = feq[k];
					}
				}
			}
		}
	}
}

__global__ void update(int NLATTICE,float*  __restrict__ d_scr,float*  __restrict__ d_dst,float*  __restrict__ d_ux,float*  __restrict__ d_uy,float*  __restrict__ d_uz,float*  __restrict__ d_rho){
	float fnq[19],feq;
	int i,j,k,ind,ind2,idx,koff,geo_tmp;
	int bx=1+(NX-1)/BLOCK_X,by=1+(NY-1)/BLOCK_Y;
	float tmp_rho,tmp_ux, tmp_uy, tmp_uz;
	float tau=0.55f;
	
	for(koff=BLOCK_Z-1;koff>=0;koff--){
		i=threadIdx.x+blockIdx.x*blockDim.x;
		j=threadIdx.y+blockIdx.y*blockDim.y;
		k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
		ind=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
		idx=tex1Dfetch(d_index,ind);
		
		geo_tmp=tex1Dfetch(d_geo,ind);
		if (geo_tmp ==4) {
			fnq[0] = d_scr[idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[1] = d_scr[NLATTICE*1+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[2] = d_scr[NLATTICE*2+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[3] = d_scr[NLATTICE*3+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y+1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[4] = d_scr[NLATTICE*4+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[5] = d_scr[NLATTICE*5+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[6] = d_scr[NLATTICE*6+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[7] = d_scr[NLATTICE*7+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=threadIdx.y+blockIdx.y*blockDim.y+1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[8] = d_scr[NLATTICE*8+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[9] = d_scr[NLATTICE*9+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=threadIdx.y+blockIdx.y*blockDim.y+1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[10] = d_scr[NLATTICE*10+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[11] = d_scr[NLATTICE*11+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[12] = d_scr[NLATTICE*12+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[13] = d_scr[NLATTICE*13+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[14] = d_scr[NLATTICE*14+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[15] = d_scr[NLATTICE*15+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y+1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[16] = d_scr[NLATTICE*16+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[17] = d_scr[NLATTICE*17+idx];
				
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y+1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[18] = d_scr[NLATTICE*18+idx];

			
			tmp_rho = 0.f;
			for (int k = 0; k < Q; k++) {
				tmp_rho = tmp_rho + fnq[k];
			}

			tmp_ux = (fnq[1] - fnq[2] + fnq[7] + fnq[8] - fnq[9] - fnq[10] + fnq[11] + fnq[12] - fnq[13] - fnq[14]) / tmp_rho;
			tmp_uy = (fnq[3] - fnq[4] + fnq[7] - fnq[8] + fnq[9] - fnq[10] + fnq[15] - fnq[16] + fnq[17] - fnq[18]) / tmp_rho;
			tmp_uz = (fnq[5] - fnq[6] + fnq[11] - fnq[12] + fnq[13] - fnq[14] + fnq[15] + fnq[16] - fnq[17] - fnq[18]) / tmp_rho;

			idx=tex1Dfetch(d_index,ind);
			/**
			if(geo_tmp==5){
				k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
				ind2=i + k*NX;
				tmp_ux=tex1Dfetch(d_meax,ind2);
				tmp_uy=tex1Dfetch(d_meay,ind2);
			}
			**/
			d_rho[idx] = tmp_rho;			
			d_ux[idx] = tmp_ux;
			d_uy[idx] = tmp_uy;
			d_uz[idx] = tmp_uz;			

			feq = tmp_rho/3.0f * (1.0f - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uy*tmp_uy -1.5f* tmp_uz*tmp_uz);
			d_dst[idx] = fnq[0] - (fnq[0] - feq) / tau;
			feq = tmp_rho /18.0f * (1.0f + 3.0f* tmp_ux + 3.0f*tmp_ux*tmp_ux -1.5f* tmp_uy*tmp_uy -1.5f* tmp_uz*tmp_uz);
			d_dst[NLATTICE*1+idx] = fnq[1] - (fnq[1] - feq) / tau;
			feq = tmp_rho /18.0f * (1.0f - 3.0f* tmp_ux + 3.0f*tmp_ux*tmp_ux -1.5f* tmp_uy*tmp_uy -1.5f* tmp_uz*tmp_uz);
			d_dst[NLATTICE*2+idx] = fnq[2] - (fnq[2] - feq) / tau;
			feq = tmp_rho /18.0f * (1.0f + 3.0f* tmp_uy + 3.0f*tmp_uy*tmp_uy - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uz*tmp_uz);
			d_dst[NLATTICE*3+idx] = fnq[3] - (fnq[3] - feq) / tau;
			feq = tmp_rho /18.0f * (1.0f - 3.0f* tmp_uy + 3.0f*tmp_uy*tmp_uy - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uz*tmp_uz);
			d_dst[NLATTICE*4+idx] = fnq[4] - (fnq[4] - feq) / tau;
			feq = tmp_rho /18.0f * (1.0f + 3.0f* tmp_uz + 3.0f*tmp_uz*tmp_uz - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uy*tmp_uy);
			d_dst[NLATTICE*5+idx] = fnq[5] - (fnq[5] - feq) / tau;
			feq = tmp_rho /18.0f* (1.0f - 3.0f* tmp_uz + 3.0f*tmp_uz*tmp_uz - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uy*tmp_uy);
			d_dst[NLATTICE*6+idx] = fnq[6] - (fnq[6] - feq) / tau;
			feq = tmp_rho /36.0f* (1.0f + 3.0f* (tmp_ux + tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy + 9.0f*tmp_ux*tmp_uy -1.5f* tmp_uz*tmp_uz);
			d_dst[NLATTICE*7+idx] = fnq[7] - (fnq[7] - feq) / tau;
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_ux - tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy - 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
			d_dst[NLATTICE*8+idx] = fnq[8] - (fnq[8] - feq) / tau;
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uy - tmp_ux) + 3.0f* tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy - 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
			d_dst[NLATTICE*9+idx] = fnq[9] - (fnq[9] - feq) / tau;
			feq = tmp_rho /36.0f * (1.0f - 3.0f* (tmp_ux + tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy + 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
			d_dst[NLATTICE*10+idx] = fnq[10] - (fnq[10] - feq) / tau;
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_ux + tmp_uz) + 3.0f* tmp_ux*tmp_ux + 3.0f*tmp_uz*tmp_uz + 9.0f*tmp_ux*tmp_uz-1.5f* tmp_uy*tmp_uy);
			d_dst[NLATTICE*11+idx] = fnq[11] - (fnq[11] - feq) / tau;
			feq = tmp_rho /36.0f* (1.0f + 3.0f* (tmp_ux - tmp_uz) + 3.0f* tmp_ux*tmp_ux + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_ux*tmp_uz-1.5f* tmp_uy*tmp_uy);
			d_dst[NLATTICE*12+idx] = fnq[12] - (fnq[12] - feq) / tau;
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uz - tmp_ux) + 3.0f* tmp_ux*tmp_ux + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_ux*tmp_uz-1.5f* tmp_uy*tmp_uy);
			d_dst[NLATTICE*13+idx] = fnq[13] - (fnq[13] - feq) / tau;
			feq = tmp_rho /36.0f * (1.0f - 3.0f* (tmp_ux + tmp_uz) + 3.0f* tmp_ux*tmp_ux + 3.0*tmp_uz*tmp_uz + 9.0f*tmp_ux*tmp_uz -1.5f* tmp_uy*tmp_uy);
			d_dst[NLATTICE*14+idx] = fnq[14] - (fnq[14] - feq) / tau;
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uy + tmp_uz) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz + 9.0f*tmp_uy*tmp_uz- 1.5f*tmp_ux*tmp_ux);
			d_dst[NLATTICE*15+idx] = fnq[15] - (fnq[15] - feq) / tau;
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uz - tmp_uy) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
			d_dst[NLATTICE*16+idx] = fnq[16] - (fnq[16] - feq) / tau;
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uy - tmp_uz) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
			d_dst[NLATTICE*17+idx] = fnq[17] - (fnq[17] - feq) / tau;
			feq = tmp_rho /36.0f * (1.0f - 3.0f* (tmp_uy + tmp_uz) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz + 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
			d_dst[NLATTICE*18+idx] = fnq[18] - (fnq[18] - feq) / tau;
		}
	}
}

__global__ void boundary_stream(int NLATTICE,float*  __restrict__ d_scr,float*  __restrict__ d_dst,float*  __restrict__ d_ux,float*  __restrict__ d_uy,float*  __restrict__ d_uz,float*  __restrict__ d_rho){
	float feq,fnq[19];
	int i,j,k,ind,idx,ind2,idx2,koff,geo_tmp;
	int bx=1+(NX-1)/BLOCK_X,by=1+(NY-1)/BLOCK_Y;
	float tmp, tmp_rho,tmp_ux, tmp_uy, tmp_uz,uygt_in,uygt_out,tau=0.55f;
	
	for(koff=BLOCK_Z-1;koff>=0;koff--){
		i=threadIdx.x+blockIdx.x*blockDim.x;
		j=threadIdx.y+blockIdx.y*blockDim.y;
		k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
		ind=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
		ind2=i + k*NX;
		geo_tmp=tex1Dfetch(d_geo,ind);
		idx2=tex1Dfetch(d_index,ind);
		
		if (geo_tmp == 1) {
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[1] = d_dst[NLATTICE+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[2] = d_dst[NLATTICE*2+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=(threadIdx.y+blockIdx.y*blockDim.y-1+NY)%NY;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[3] = d_dst[NLATTICE*3+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=(threadIdx.y+blockIdx.y*blockDim.y+1)%NY;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[4] = d_dst[NLATTICE*4+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[5] = d_dst[NLATTICE*5+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[6] = d_dst[NLATTICE*6+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=(threadIdx.y+blockIdx.y*blockDim.y-1+NY)%NY;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[7] = d_dst[NLATTICE*7+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=(threadIdx.y+blockIdx.y*blockDim.y+1)%NY;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[8] = d_dst[NLATTICE*8+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=(threadIdx.y+blockIdx.y*blockDim.y-1+NY)%NY;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[9] = d_dst[NLATTICE*9+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=(threadIdx.y+blockIdx.y*blockDim.y+1)%NY;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[10] = d_dst[NLATTICE*10+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[11] = d_dst[NLATTICE*11+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[12] = d_dst[NLATTICE*12+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[13] = d_dst[NLATTICE*13+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=threadIdx.y+blockIdx.y*blockDim.y;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[14] = d_dst[NLATTICE*14+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=(threadIdx.y+blockIdx.y*blockDim.y-1+NY)%NY;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[15] = d_dst[NLATTICE*15+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=(threadIdx.y+blockIdx.y*blockDim.y+1)%NY;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[16] = d_dst[NLATTICE*16+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=(threadIdx.y+blockIdx.y*blockDim.y-1+NY)%NY;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[17] = d_dst[NLATTICE*17+idx];
			
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=(threadIdx.y+blockIdx.y*blockDim.y+1)%NY;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			fnq[18] = d_dst[NLATTICE*18+idx];
			
			d_dst[NLATTICE*1+idx2] = fnq[2];
			d_dst[NLATTICE*2+idx2] = fnq[1];
			d_dst[NLATTICE*3+idx2] = fnq[4];
			d_dst[NLATTICE*4+idx2] = fnq[3];
			d_dst[NLATTICE*5+idx2] = fnq[6];
			d_dst[NLATTICE*6+idx2] = fnq[5];
			d_dst[NLATTICE*7+idx2] = fnq[10];
			d_dst[NLATTICE*8+idx2] = fnq[9];
			d_dst[NLATTICE*9+idx2] = fnq[8];
			d_dst[NLATTICE*10+idx2] = fnq[7];
			d_dst[NLATTICE*11+idx2] = fnq[14];
			d_dst[NLATTICE*12+idx2] = fnq[13];
			d_dst[NLATTICE*13+idx2] = fnq[12];
			d_dst[NLATTICE*14+idx2] = fnq[11];
			d_dst[NLATTICE*15+idx2] = fnq[18];
			d_dst[NLATTICE*16+idx2] = fnq[17];
			d_dst[NLATTICE*17+idx2] = fnq[16];
			d_dst[NLATTICE*18+idx2] = fnq[15];
		}
		
		/**
		if (geo_tmp == 3) {
			uygt_out=tex1Dfetch(d_outlety,ind2);
			//4
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
						
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /18.0f * (1.0f - 3.0f* tmp_uy + 3.0f*tmp_uy*tmp_uy - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uz*tmp_uz);
			tmp = tmp_rho /18.0f * (1.0f - 3.0f* uygt_out + 3.0f*uygt_out*uygt_out);
			d_dst[NLATTICE*4+idx2] = tmp + (d_dst[NLATTICE*4+idx] - feq)*(1.0f - 1.0f / tau);

			//8
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_ux - tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy - 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
			tmp = tmp_rho /36.0f * (1.0f - 3.0f* uygt_out+ 3.0f*uygt_out*uygt_out);
			d_dst[NLATTICE*8+idx2] = tmp + (d_dst[NLATTICE*8+idx] - feq)*(1.0f - 1.0f / tau);

			//10
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f * (1.0f - 3.0f* (tmp_ux + tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy + 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
			tmp = tmp_rho /36.0f * (1.0f - 3.0f*uygt_out + 3.0f*uygt_out*uygt_out);
			d_dst[NLATTICE*10+idx2] = tmp + (d_dst[NLATTICE*10+idx] - feq)*(1.0f - 1.0f / tau);

			//16
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uz - tmp_uy) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
			tmp = tmp_rho /36.0f * (1.0f - 3.0f* uygt_out + 3.0f* uygt_out*uygt_out);
			d_dst[NLATTICE*16+idx2] = tmp + (d_dst[NLATTICE*16+idx] - feq)*(1.0f - 1.0f / tau);
			
			//18
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f * (1.0f - 3.0f* (tmp_uy + tmp_uz) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz + 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
			tmp = tmp_rho /36.0f * (1.0f - 3.0f* uygt_out + 3.0f* uygt_out*uygt_out );
			d_dst[NLATTICE*18+idx2] = tmp + (d_dst[NLATTICE*18+idx] - feq)*(1.0f - 1.0f / tau);
		}
		**/
		
		if (geo_tmp == 3) {
			//4
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
						
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /18.0f * (1.0f - 3.0f* tmp_uy + 3.0f*tmp_uy*tmp_uy - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uz*tmp_uz);
			tmp = 1.f /18.0f * (1.0f - 3.0f* tmp_uy + 3.0f*tmp_uy*tmp_uy - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uz*tmp_uz);
			d_dst[NLATTICE*4+idx2] = tmp + (d_dst[NLATTICE*4+idx] - feq)*(1.0f - 1.0f / tau);

			//8
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_ux - tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy - 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
			tmp = 1.f /36.0f * (1.0f + 3.0f* (tmp_ux - tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy - 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
			d_dst[NLATTICE*8+idx2] = tmp + (d_dst[NLATTICE*8+idx] - feq)*(1.0f - 1.0f / tau);

			//10
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f * (1.0f - 3.0f* (tmp_ux + tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy + 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
			tmp = 1.f /36.0f * (1.0f - 3.0f* (tmp_ux + tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy + 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
			d_dst[NLATTICE*10+idx2] = tmp + (d_dst[NLATTICE*10+idx] - feq)*(1.0f - 1.0f / tau);

			//16
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uz - tmp_uy) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
			tmp = 1.f /36.0f * (1.0f + 3.0f* (tmp_uz - tmp_uy) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
			d_dst[NLATTICE*16+idx2] = tmp + (d_dst[NLATTICE*16+idx] - feq)*(1.0f - 1.0f / tau);
			
			//18
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y-1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f * (1.0f - 3.0f* (tmp_uy + tmp_uz) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz + 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
			tmp = 1.f /36.0f * (1.0f - 3.0f* (tmp_uy + tmp_uz) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz + 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
			d_dst[NLATTICE*18+idx2] = tmp + (d_dst[NLATTICE*18+idx] - feq)*(1.0f - 1.0f / tau);
		}
		
		if (geo_tmp == 2) {
			uygt_in=tex1Dfetch(d_inlety,ind2);
			//3
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y+1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /18.0f * (1.0f + 3.0f* tmp_uy + 3.0f*tmp_uy*tmp_uy - 1.5f*tmp_ux*tmp_ux -1.5f* tmp_uz*tmp_uz);
			tmp = tmp_rho /18.0f * (1.0f + 3.0f* uygt_in + 3.0f*uygt_in*uygt_in);
			d_dst[NLATTICE*3+idx2] = tmp + (d_dst[NLATTICE*3+idx] - feq)*(1.0f - 1.0f / tau);

			//7
			i=threadIdx.x+blockIdx.x*blockDim.x+1;
			j=threadIdx.y+blockIdx.y*blockDim.y+1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f* (1.0f + 3.0f* (tmp_ux + tmp_uy) + 3.0f*tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy + 9.0f*tmp_ux*tmp_uy -1.5f* tmp_uz*tmp_uz);
			tmp = tmp_rho /36.0f* (1.0f + 3.0f* uygt_in + 3.0f*uygt_in*uygt_in);
			d_dst[NLATTICE*7+idx2] = tmp + (d_dst[NLATTICE*7+idx] - feq)*(1.0f - 1.0f / tau);

			//9
			i=threadIdx.x+blockIdx.x*blockDim.x-1;
			j=threadIdx.y+blockIdx.y*blockDim.y+1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uy - tmp_ux) + 3.0f* tmp_ux*tmp_ux + 3.0f*tmp_uy*tmp_uy - 9.0f*tmp_ux*tmp_uy-1.5f* tmp_uz*tmp_uz);
			tmp = tmp_rho /36.0f * (1.0f + 3.0f* uygt_in+ 3.0f*uygt_in*uygt_in);
			d_dst[NLATTICE*9+idx2] = tmp + (d_dst[NLATTICE*9+idx] - feq)*(1.0f - 1.0f / tau);
							
			//15
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y+1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff+1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uy + tmp_uz) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz + 9.0f*tmp_uy*tmp_uz- 1.5f*tmp_ux*tmp_ux);
			tmp = tmp_rho /36.0f * (1.0f + 3.0f* uygt_in + 3.0f* uygt_in*uygt_in);
			d_dst[NLATTICE*15+idx2] = tmp + (d_dst[NLATTICE*15+idx] - feq)*(1.0f - 1.0f / tau);

			//17
			i=threadIdx.x+blockIdx.x*blockDim.x;
			j=threadIdx.y+blockIdx.y*blockDim.y+1;
			k=threadIdx.z+blockIdx.z*BLOCK_Z+koff-1;
			ind2=(i/BLOCK_X+j/BLOCK_Y*bx+k/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+i%BLOCK_X+j%BLOCK_Y*BLOCK_X+k%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=tex1Dfetch(d_index,ind2);
			tmp_rho = d_rho[idx];
			tmp_ux = d_ux[idx];
			tmp_uy = d_uy[idx];
			tmp_uz = d_uz[idx];
			feq = tmp_rho /36.0f * (1.0f + 3.0f* (tmp_uy - tmp_uz) + 3.0f* tmp_uy*tmp_uy + 3.0f*tmp_uz*tmp_uz - 9.0f*tmp_uy*tmp_uz - 1.5f*tmp_ux*tmp_ux);
			tmp = tmp_rho /36.0f * (1.0f + 3.0f* uygt_in + 3.0f* uygt_in*uygt_in );
			d_dst[NLATTICE*17+idx2] = tmp + (d_dst[NLATTICE*17+idx] - feq)*(1.0f - 1.0f / tau);
		}
	}
}
			
void outtxt(){
	int ind,idx;
	int z=NZ/2;
	ofstream ofs("s1_out.txt");	
	for (int y = 0; y < NY; y++) {
		for (int x = 0; x < NX; x++) {
			ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			if (h_geo[ind] == 4) {
				idx=h_index[ind];
				ofs<<h_uy[idx]<<' ';
			}else{
				ofs<<0<<' ';
			}
		}
	}
	
	for (int y = 0; y < NY; y++) {
		for (int x = 0; x < NX; x++) {
			ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			if (h_geo[ind] == 4) {
				idx=h_index[ind];
				ofs<<h_ux[idx]<<' ';
			}else{
				ofs<<0<<' ';
			}
		}
	}
	ofs.close();
}

void write_once() {
	int ind, idx, z = NZ/2;
	ofstream ofs("./meas1.txt");	
	for (int y = 0; y < NY; y++) {
		for (int x = 0; x < NX; x++) {
			ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=h_index[ind];
			ofs<<h_uy[idx]<<' ';
		}
	}
	
	for (int y = 0; y < NY; y++) {
		for (int x = 0; x < NX; x++) {
			ind = (x / BLOCK_X + y / BLOCK_Y*bx + z / BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z + x%BLOCK_X + y%BLOCK_Y*BLOCK_X + z%BLOCK_Z*BLOCK_X*BLOCK_Y;
			idx=h_index[ind];
			ofs<<h_ux[idx]<<' ';
		}
	}
	ofs.close();
}

void write_vel() {
	int ind,idx;
	string velo_file = "./scenario3a.txt";
	ofstream velfile(velo_file);
	
	for (int z=0; z < NZ; z++) {
		for (int y=0; y < NY; y++) {
			for (int x=0; x < NX; x++) {
				ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				idx=h_index[ind];					
				if(idx>=0)velfile<<h_ux[idx]<<' '<<h_uy[idx] <<' '<<h_uz[idx] <<' ';
				//else velfile<<0<<' '<<0 <<' '<<0 <<' ';
			}
		}
		velfile<<endl;
	}
	velfile.close();
}

void outputSave(int t) {
	int ind,idx;
	string datafilename=output_direc + "/bif_" +to_string(t)+".vtk";
	ofstream ofs(datafilename);
	ofs<<"# vtk DataFile Version 2.0"<<endl;
	ofs<<"<-- LBM flow with UIV acceleration, http://www.bg.ic.ac.uk/research/m.tang/ulis/ -->"<<endl;
	ofs<<"ASCII"<<endl;
	ofs<<"DATASET STRUCTURED_POINTS"<<endl;
	ofs << "DIMENSIONS " << NX-2 << ' ' << NY-4 << ' ' << NZ-2 << endl;
	ofs<< "SPACING "<< CH<<' '<< CH<<' '<< CH<<endl;
	ofs<<"ORIGIN "<< round(NX / 2)*CH<<' '<< round(NY / 2)*CH<<' '<< .0<<endl;
	ofs<<"POINT_DATA  "<<(NX-2) *(NY-4)*(NZ-2)<<endl;
/**
	ofs << "SCALARS DENSITY float" << endl;
	ofs << "LOOKUP_TABLE default" << endl;
	for (int z=1; z < NZ-1; z++) {
		for (int y=2; y < NY-2; y++) {
			for (int x=1; x < NX-1; x++) {
				ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				idx=h_index[ind];
				if(idx>=0)ofs << h_rho[idx] * C_rho << ' ';
				else ofs << 0.0f<< ' ';
			}
		}
	}
	ofs << endl;

	ofs << "SCALARS PRESSURE float" << endl;
	ofs << "LOOKUP_TABLE default" << endl;
	for (int z=1; z < NZ-1; z++) {
		for (int y=2; y < NY-2; y++) {
			for (int x=1; x < NX-1; x++) {
				ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				idx=h_index[ind];	
				if(idx>=0)ofs << h_rho[idx] * C_pre / 3.0 << ' ';
				else ofs << 0.0f<< ' ';
			}
		}
	}
	ofs << endl;
**/
	ofs << "VECTORS VELOCITY float" << endl;
	for (int z=1; z < NZ-1; z++) {
		for (int y=2; y < NY-2; y++) {
			for (int x=1; x < NX-1; x++) {
				ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				idx=h_index[ind];
				if(idx>=0){
					ofs << h_ux[idx] * C_U << ' ';
					ofs << h_uy[idx] * C_U << ' ';
					ofs << h_uz[idx] * C_U << ' ';
				}else{
					ofs << 0 << ' ';
					ofs << 0 << ' ';
					ofs << 0 << ' ';
				}
			}
		}
	}

	ofs.close();
}

long double calc_res() {
	int ind,idx;
	float vtmp;
	long double sum1=0.0L;
	for (int z=1; z < NZ-1; z++) {
		for (int y=2; y < NY-2; y++) {
			for (int x=1; x < NX-1; x++) {
				ind=(x/BLOCK_X+y/BLOCK_Y*bx+z/BLOCK_Z*bx*by)*BLOCK_X*BLOCK_Y*BLOCK_Z+x%BLOCK_X+y%BLOCK_Y*BLOCK_X+z%BLOCK_Z*BLOCK_X*BLOCK_Y;
				idx=h_index[ind];
				if(h_geo[ind]>=4){
					vtmp = powf(h_ux[idx], 2.f) + powf(h_uy[idx], 2.f) + powf(h_uz[idx], 2.f);
					sum1 = sum1 + vtmp;				
				}
			}
		}
	}
	return sum1;
}

int main(int argc,const char **argv) {
	float residual,milli;
	long double sum1,sum2;
	logname=output_direc+'/'+ "CONVERGENCE.log";
	ofstream logfile(logname);	
	dim3 dimGrid(bx,by,bz);
	dim3 dimBlock(BLOCK_X,BLOCK_Y,1);

	h_index=(int*)malloc(sizeof(int)*bx*by*bz*BLOCK_X*BLOCK_Y*BLOCK_Z);
	h_geo=(int*)malloc(sizeof(int)*bx*by*bz*BLOCK_X*BLOCK_Y*BLOCK_Z);
	geo_pre();
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);	

	
	//allocate memory
	h_ux=(float*)malloc(sizeof(float)*NLATTICE);
	h_uy=(float*)malloc(sizeof(float)*NLATTICE);
	h_uz=(float*)malloc(sizeof(float)*NLATTICE);
	h_rho=(float*)malloc(sizeof(float)*NLATTICE);
	h_meax=(float*)malloc(sizeof(float)*NX*NY);
	h_meay=(float*)malloc(sizeof(float)*NX*NY);
	h_inletx = (float*)malloc(sizeof(float)*NX*NZ);
	h_inlety = (float*)malloc(sizeof(float)*NX*NZ);
	h_outletx = (float*)malloc(sizeof(float)*NX*NZ);
	h_outlety = (float*)malloc(sizeof(float)*NX*NZ);
	h_dst=(float*)malloc(sizeof(float)*NLATTICE*Q);
	h_scr=(float*)malloc(sizeof(float)*NLATTICE*Q);
	
	hipMalloc((void**)&d_indexdev,sizeof(int)*bx*by*bz*BLOCK_X*BLOCK_Y*BLOCK_Z);
	hipMalloc((void**)&d_ux,sizeof(float)*NLATTICE);
	hipMalloc((void**)&d_uy,sizeof(float)*NLATTICE);
	hipMalloc((void**)&d_uz,sizeof(float)*NLATTICE);
	hipMalloc((void**)&d_rho,sizeof(float)*NLATTICE);	
	
	hipMalloc(&d_meax1,NX*NY*sizeof(float));
	hipMalloc(&d_meay1,NX*NY*sizeof(float));
	hipMalloc(&d_inletx1,NX*NZ*sizeof(float));
	hipMalloc(&d_inlety1,NX*NZ*sizeof(float));
	hipMalloc(&d_outletx1,NX*NZ*sizeof(float));
	hipMalloc(&d_outlety1,NX*NZ*sizeof(float));	
	hipMalloc(&d_geo1,bx*by*bz*BLOCK_X*BLOCK_Y*BLOCK_Z*sizeof(int));

	hipMalloc((void**)&d_dst,sizeof(float)*NLATTICE*Q);
	hipMalloc((void**)&d_scr,sizeof(float)*NLATTICE*Q);
	read_vel();
	initialize();
	hipMemcpy(d_indexdev,h_index,sizeof(int)*bx*by*bz*BLOCK_X*BLOCK_Y*BLOCK_Z,hipMemcpyHostToDevice);
 	hipMemcpy(d_meax1,h_meax,NX*NY*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_meay1,h_meay,NX*NY*sizeof(float),hipMemcpyHostToDevice);
 	hipMemcpy(d_inletx1,h_inletx,NX*NZ*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_inlety1,h_inlety,NX*NZ*sizeof(float),hipMemcpyHostToDevice);
 	hipMemcpy(d_outletx1,h_outletx,NX*NZ*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_outlety1,h_outlety,NX*NZ*sizeof(float),hipMemcpyHostToDevice);	
	hipMemcpy(d_geo1,h_geo,bx*by*bz*BLOCK_X*BLOCK_Y*BLOCK_Z*sizeof(int),hipMemcpyHostToDevice);
	
	hipMemcpy(d_dst,h_dst,sizeof(float)*NLATTICE*Q,hipMemcpyHostToDevice);
	hipMemcpy(d_scr,h_scr,sizeof(float)*NLATTICE*Q,hipMemcpyHostToDevice);

	hipBindTexture(NULL,d_index,d_indexdev);	
	hipBindTexture(NULL,d_meax,d_meax1);
	hipBindTexture(NULL,d_meay,d_meay1);
	hipBindTexture(NULL,d_inletx,d_inletx1);
	hipBindTexture(NULL,d_inlety,d_inlety1);
	hipBindTexture(NULL,d_outletx,d_outletx1);
	hipBindTexture(NULL,d_outlety,d_outlety1);	
	hipBindTexture(NULL,d_geo,d_geo1);
	hipEventRecord(start);
	for(int i=0;i<=REPEAT;i++){
		hipDeviceSynchronize();
	
		update<<<dimGrid,dimBlock>>>(NLATTICE,d_scr,d_dst,d_ux,d_uy,d_uz,d_rho);
		hipDeviceSynchronize();

		boundary_stream<<<dimGrid,dimBlock>>>(NLATTICE,d_scr,d_dst,d_ux,d_uy,d_uz,d_rho);
		hipDeviceSynchronize();
		
		d_tmp=d_scr;
		d_scr = d_dst;
		d_dst=d_tmp;
		if(i%time_save==0){
			hipDeviceSynchronize();
			sum1=calc_res();
			hipMemcpy(h_ux,d_ux,sizeof(float)*NLATTICE,hipMemcpyDeviceToHost);
			hipMemcpy(h_uy,d_uy,sizeof(float)*NLATTICE,hipMemcpyDeviceToHost);
			hipMemcpy(h_uz,d_uz,sizeof(float)*NLATTICE,hipMemcpyDeviceToHost);
			hipMemcpy(h_rho,d_rho,sizeof(float)*NLATTICE,hipMemcpyDeviceToHost);			
			hipEventRecord(stop);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&milli,start,stop);
			sum2=calc_res();
			residual=(float)(abs(sum1-sum2)/sum2);
			logfile<<residual<<endl;
			cout << "ITERATION # " << i << ", collapse time: " << milli <<" ms, residual:" << residual << endl;
			outputSave(i);
		}
	}
	//outtxt();
	//write_vel();
	write_once();
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milli,start,stop);
	cout << "TOTAL RUNNING TIME: " << milli << " MILLI SECONDS" << "#LATTICE" << NLATTICE << endl;
	logfile << "TOTAL RUNNING TIME: " << milli << " MILLI SECONDS" << "#LATTICE" << NLATTICE << " ERROR IS"<<residual<<endl;
	logfile.close();
	
	hipUnbindTexture(d_index);	
	hipUnbindTexture(d_geo);
	hipUnbindTexture(d_meax);
	hipUnbindTexture(d_meay);
	hipUnbindTexture(d_inletx);
	hipUnbindTexture(d_inlety);
	hipUnbindTexture(d_outletx);
	hipUnbindTexture(d_outlety);	
	//free memory
	free(h_index);
	free(h_ux);
	free(h_uy);
	free(h_uz);
	free(h_rho);
	free(h_meax);
	free(h_meay);
	free(h_dst);
	free(h_scr);
	free(h_geo);
	free(h_inletx);
	free(h_inlety);
	free(h_outletx);
	free(h_outlety);
	
	hipFree(d_indexdev);
	hipFree(d_dst);
	hipFree(d_scr);
	hipFree(d_geo1);
	hipFree(d_ux);
	hipFree(d_uy);
	hipFree(d_uz);
	hipFree(d_rho);	
	hipFree(d_meax1);
	hipFree(d_meay1);
	hipFree(d_inletx1);
	hipFree(d_inlety1);
	hipFree(d_outletx1);
	hipFree(d_outlety1);	
	hipDeviceReset();
	system("pause");
	return 0;
}
